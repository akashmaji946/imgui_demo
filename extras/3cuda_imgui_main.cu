#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath>

// ImGui + GLFW + OpenGL3
#include "../imgui/imgui.h"
#include "../backends/imgui_impl_glfw.h"
#include "../backends/imgui_impl_opengl3.h"
#include "../common/OFFReader.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../common/stb_image_write.h"

#define GL_SILENCE_DEPRECATION
#if defined(IMGUI_IMPL_OPENGL_ES2)
#include <GLES2/gl2.h>
#else
#include <GL/gl.h>
#endif
#include <GLFW/glfw3.h>

// Project includes
#include "../include/Renderer.hpp"
#include "../include/Point.hpp"
#include "../include/Vector.hpp"
#include "../include/Camera.hpp"
#include "../include/Scene.hpp"

// Forward declare your kernels from the original snippet
__global__ void render(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);
__global__ void render_antialias(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);
__global__ void render_cool(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);

static void glfw_error_callback(int error, const char* description) {
    std::fprintf(stderr, "GLFW Error %d: %s\n", error, description);
}

// Global camera state
static double camera_vfov = 20.0;
static Point camera_lookat = Point(0, 0, -1);
static Point camera_lookfrom = Point(0, 0, 0); 
static bool camera_changed = true;

// Mouse tracking variables are no longer needed
static float mouse_x = 90.0f; 
static float mouse_y = 0.0f;
static float camera_radius = 10.0f;

// Mouse callbacks are no longer needed
void mouse_callback(GLFWwindow* window, double xpos, double ypos) {}
void mouse_button_callback(GLFWwindow* window, int button, int action, int mods) {}
void scroll_callback(GLFWwindow* window, double xoffset, double yoffset) {}

int main(int argc, char** argv) {
    // Window + GL
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit()) return 1;

#if defined(IMGUI_IMPL_OPENGL_ES2)
    const char* glsl_version = "#version 100";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    glfwWindowHint(GLFW_CLIENT_API, GLFW_OPENGL_ES_API);
#elif defined(__APPLE__)
    const char* glsl_version = "#version 150";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#else
    const char* glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
#endif

    GLFWwindow* window = glfwCreateWindow(1920, 1080, "CUDA ImGui Viewer", nullptr, nullptr);
    if (!window) return 1;
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);

    // Mouse callbacks are commented out as they are no longer needed
    // glfwSetCursorPosCallback(window, mouse_callback);
    // glfwSetMouseButtonCallback(window, mouse_button_callback);
    // glfwSetScrollCallback(window, scroll_callback);

    // ImGui setup
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;
    ImGui::StyleColorsDark();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);

    // Renderer setup
    int image_width = 640;
    int image_height = 480;
    int samples_per_pixel = 1;
    int kernel_choice = 2;

    // char filename[] = "teapot.OFF";
    // char* filename = argv[1];
    OFFModel* model = readOffFile(argv[1]);
    if(model == NULL) {
        std::cerr << "Failed to read OFF file: " << argv[1] << std::endl;
        return 1;
    }

    Renderer renderer(image_width, image_height, samples_per_pixel, model);
    
    FreeOffModel(model);

    // Initial render setup
    renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
    renderer.render_frame(kernel_choice);

    // FPS tracking variables
    double last_time = glfwGetTime();
    int frame_count = 0;
    double fps = 0.0;

    // Main loop
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();
        
        // FPS calculation
        double current_time = glfwGetTime();
        frame_count++;
        if (current_time - last_time >= 1.0) {
            fps = static_cast<double>(frame_count) / (current_time - last_time);
            frame_count = 0;
            last_time = current_time;
        }

        // Check for camera movement and update renderer
        if (camera_changed) {
            // Recalculate camera_lookfrom based on current yaw, pitch, and radius
            double pitch_rad = mouse_y * M_PI / 180.0;
            double yaw_rad = mouse_x * M_PI / 180.0;

            camera_lookfrom.m_x = camera_lookat.m_x + camera_radius * cos(yaw_rad) * cos(pitch_rad);
            camera_lookfrom.m_y = camera_lookat.m_y + camera_radius * sin(pitch_rad);
            camera_lookfrom.m_z = camera_lookat.m_z + camera_radius * sin(yaw_rad) * cos(pitch_rad);

            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            camera_changed = false;
            renderer.render_frame(kernel_choice);
        }

        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // Controls
        ImGui::Begin("Controls");
        ImGui::Text("CUDA ImGui Viewer");
        ImGui::Separator();
        ImGui::Text("FPS: %.2f", fps);
        ImGui::Separator();
        
        bool changed_local = false;
        changed_local |= ImGui::SliderFloat("Yaw", &mouse_x, -360.0f, 360.0f);
        changed_local |= ImGui::SliderFloat("Pitch", &mouse_y, -89.0f, 89.0f);
        changed_local |= ImGui::SliderFloat("Zoom", &camera_radius, 0.1f, 20.0f); // New zoom slider
        changed_local |= ImGui::SliderInt("Samples Per Pixel", &samples_per_pixel, 1, 100);
        changed_local |= ImGui::SliderInt("Width", &image_width, 160, 1920);
        changed_local |= ImGui::SliderInt("Height", &image_height, 90, 1080);
        changed_local |= ImGui::SliderFloat("FOV", (float*)&camera_vfov, 5.0f, 60.0f);
        if (ImGui::Combo("Kernel", &kernel_choice, "simple\0antialias\0cool\0")) changed_local = true;
        
        // This is the key change: update the global flag.
        if (changed_local) {
            camera_changed = true;
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
        }

        if (ImGui::Button("Renders")) {
            std::cout << "Rendering..." << std::endl;
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
        }
        ImGui::SameLine();
        if (ImGui::Button("Save PNG")) {
            std::cout << "Saving image..." << std::endl;
            stbi_write_png("image.png", image_width, image_height, 3, renderer.get_host_image(), image_width * 3);
            std::cout << "Image saved!" << std::endl;
        }

        if(ImGui::Button("Reset")) {
            image_width = 1080;
            image_height = 720;
            
            samples_per_pixel = 1;
            kernel_choice = 2;
            camera_radius = 10;
            
            // Reset mouse and camera values
            mouse_x = 90.0f;
            mouse_y = 0.0f;

            camera_lookfrom = Point(0, 0, 0);
            camera_lookat = Point(0, 0, -1);
            camera_vfov = 20.0;
            
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
            camera_changed = true;

        }

        ImGui::End();

        // Update the texture only when the CUDA kernel has finished
        if (renderer.is_ready()) {
            renderer.update_texture();
        }

        // Render to the main window
        ImGui::Render();
        int dw, dh; 
        glfwGetFramebufferSize(window, &dw, &dh);
        glViewport(0, 0, dw, dh);
        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
        
        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        glOrtho(0, dw, dh, 0, -1, 1);
        glMatrixMode(GL_MODELVIEW);
        glLoadIdentity();

        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, renderer.get_texture_id());
        
        // Corrected OpenGL rendering for aspect ratio
        float window_aspect = (float)dw / dh;
        float image_aspect = (float)image_width / image_height;
        float quad_width, quad_height;
        float x_offset, y_offset;
        
        if (window_aspect > image_aspect) {
            quad_height = (float)dh;
            quad_width = quad_height * image_aspect;
            x_offset = (dw - quad_width) * 0.5f;
            y_offset = 0;
        } else {
            quad_width = (float)dw;
            quad_height = quad_width / image_aspect;
            x_offset = 0;
            y_offset = (dh - quad_height) * 0.5f;
        }

        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 1.0f); glVertex2f(x_offset, y_offset + quad_height);
        glTexCoord2f(1.0f, 1.0f); glVertex2f(x_offset + quad_width, y_offset + quad_height);
        glTexCoord2f(1.0f, 0.0f); glVertex2f(x_offset + quad_width, y_offset);
        glTexCoord2f(0.0f, 0.0f); glVertex2f(x_offset, y_offset);
        glEnd();
        
        glDisable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, 0);

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        glfwSwapBuffers(window);

        std::cout << "Frame rendered" << std::endl;
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}