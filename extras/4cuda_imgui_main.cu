#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath>

// ImGui + GLFW + OpenGL3
#include "../imgui/imgui.h"
#include "../backends/imgui_impl_glfw.h"
#include "../backends/imgui_impl_opengl3.h"
#include "../common/OFFReader.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../common/stb_image_write.h"

#define GL_SILENCE_DEPRECATION
#if defined(IMGUI_IMPL_OPENGL_ES2)
#include <GLES2/gl2.h>
#else
#include <GL/gl.h>
#endif
#include <GLFW/glfw3.h>

// Project includes
#include "../include/Renderer.hpp"
#include "../include/Point.hpp"
#include "../include/Vector.hpp"
#include "../include/Camera.hpp"
#include "../include/Scene.hpp"

// Forward declare kernels
__global__ void render(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);
__global__ void render_antialias(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);
__global__ void render_cool(unsigned char* image, const Camera* cam, const Scene* scene, int image_width, int image_height);

static void glfw_error_callback(int error, const char* description) {
    std::fprintf(stderr, "GLFW Error %d: %s\n", error, description);
}

// ==== Global camera state ====
static double camera_vfov = 20.0;
static Point camera_lookat = Point(0, 0, -1);
static Point camera_lookfrom = Point(0, 0, 0);
static bool camera_changed = true;


// ==== Global camera state ====
static float mouse_x = 0.0f;    // yaw (start at 0 for natural orientation)
static float mouse_y = 20.0f;   // pitch (slightly above horizontal)
static float camera_radius = 10.0f; // closer default zoom

// ==== Mouse interaction globals ====
static bool left_mouse_pressed = false;
static double last_mouse_x = 0.0, last_mouse_y = 0.0;

// ==== Callbacks ====
void mouse_callback(GLFWwindow* window, double xpos, double ypos) {
    if (left_mouse_pressed) {
        float dx = static_cast<float>(xpos - last_mouse_x);
        float dy = static_cast<float>(ypos - last_mouse_y);

        float sensitivity = 0.1f;   // smoother, slower
        mouse_x -= dx * sensitivity; // invert X for natural orbit
        mouse_y -= dy * sensitivity; // invert Y for natural orbit

        // clamp pitch
        if (mouse_y > 89.0f) mouse_y = 89.0f;
        if (mouse_y < -89.0f) mouse_y = -89.0f;

        camera_changed = true;
    }
    last_mouse_x = xpos;
    last_mouse_y = ypos;
}

void mouse_button_callback(GLFWwindow* window, int button, int action, int mods) {
    if (button == GLFW_MOUSE_BUTTON_LEFT) {
        if (action == GLFW_PRESS) {
            left_mouse_pressed = true;
            glfwGetCursorPos(window, &last_mouse_x, &last_mouse_y);
        } else if (action == GLFW_RELEASE) {
            left_mouse_pressed = false;
        }
    }
}

void scroll_callback(GLFWwindow* window, double xoffset, double yoffset) {
    camera_radius -= (float)yoffset * 0.1f; // faster zoom

    if (camera_radius < 0.5f) camera_radius = 0.5f; 
    if (camera_radius > 20.0f) camera_radius = 20.0f;
    camera_changed = true;
}


int main(int argc, char** argv) {
    // Window + GL
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit()) return 1;

#if defined(IMGUI_IMPL_OPENGL_ES2)
    const char* glsl_version = "#version 100";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
    glfwWindowHint(GLFW_CLIENT_API, GLFW_OPENGL_ES_API);
#elif defined(__APPLE__)
    const char* glsl_version = "#version 150";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
#else
    const char* glsl_version = "#version 130";
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);
#endif

    GLFWwindow* window = glfwCreateWindow(1920, 1080, "CUDA ImGui Viewer", nullptr, nullptr);
    if (!window) return 1;
    glfwMakeContextCurrent(window);
    glfwSwapInterval(1);

    // Enable mouse callbacks
    glfwSetCursorPosCallback(window, mouse_callback);
    glfwSetMouseButtonCallback(window, mouse_button_callback);
    glfwSetScrollCallback(window, scroll_callback);

    // ImGui setup
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;
    ImGui::StyleColorsDark();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init(glsl_version);

    // Renderer setup
    int image_width = 640;
    int image_height = 480;
    int samples_per_pixel = 1;
    int kernel_choice = 2;

    OFFModel* model;
    if(argv[1] != NULL) {
        model = readOffFile(argv[1]);
        if(model == NULL) {
            std::cerr << "Failed to read OFF file: " << argv[1] << std::endl;
            return 1;
        }
    } else {
        model = NULL;
    }

    // Compute bounding box
    Point min_pt(1e9, 1e9, 1e9);
    Point max_pt(-1e9, -1e9, -1e9);

    for (int i = 0; i < model->numberOfVertices; i++) {
        Vector v(model->vertices[i].x, model->vertices[i].y, model->vertices[i].z);
        min_pt.m_x = std::min(min_pt.m_x, v.m_x);
        min_pt.m_y = std::min(min_pt.m_y, v.m_y);
        min_pt.m_z = std::min(min_pt.m_z, v.m_z);

        max_pt.m_x = std::max(max_pt.m_x, v.m_x);
        max_pt.m_y = std::max(max_pt.m_y, v.m_y);
        max_pt.m_z = std::max(max_pt.m_z, v.m_z);
    }

    // Center of model
    camera_lookat = Point(
        0.5 * (min_pt.m_x + max_pt.m_x),
        0.5 * (min_pt.m_y + max_pt.m_y),
        0.5 * (min_pt.m_z + max_pt.m_z)
    );

    // Estimate radius so object fits nicely
    double dx = max_pt.m_x - min_pt.m_x;
    double dy = max_pt.m_y - min_pt.m_y;
    double dz = max_pt.m_z - min_pt.m_z;
    double max_extent = std::max({dx, dy, dz});
    camera_radius = (float)(max_extent * 1.5);  // distance from center


    Renderer renderer(image_width, image_height, samples_per_pixel, model);



    // Initial render setup
    renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
    renderer.render_frame(kernel_choice);

    // FPS tracking
    double last_time = glfwGetTime();
    int frame_count = 0;
    double fps = 0.0;

    // Main loop
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        // FPS calculation
        double current_time = glfwGetTime();
        frame_count++;
        if (current_time - last_time >= 1.0) {
            fps = static_cast<double>(frame_count) / (current_time - last_time);
            frame_count = 0;
            last_time = current_time;
        }

        // Update camera if changed
        if (camera_changed) {
            double pitch_rad = mouse_y * M_PI / 180.0;
            double yaw_rad   = mouse_x * M_PI / 180.0;

            camera_lookfrom.m_x = camera_lookat.m_x + camera_radius * cos(yaw_rad) * cos(pitch_rad);
            camera_lookfrom.m_y = camera_lookat.m_y + camera_radius * sin(pitch_rad);
            camera_lookfrom.m_z = camera_lookat.m_z + camera_radius * sin(yaw_rad) * cos(pitch_rad);

            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
            camera_changed = false;
        }

        // ImGui
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::Begin("Controls");
        ImGui::Text("CUDA ImGui Viewer");
        ImGui::Separator();
        ImGui::Text("FPS: %.2f", fps);
        ImGui::Separator();

        bool changed_local = false;
        changed_local |= ImGui::SliderFloat("Yaw", &mouse_x, -360.0f, 360.0f);
        changed_local |= ImGui::SliderFloat("Pitch", &mouse_y, -89.0f, 89.0f);
        changed_local |= ImGui::SliderFloat("Zoom", &camera_radius, -1000.0f, 1000.0f);
        changed_local |= ImGui::SliderInt("Samples Per Pixel", &samples_per_pixel, 1, 100);
        changed_local |= ImGui::SliderInt("Width", &image_width, 160, 1920);
        changed_local |= ImGui::SliderInt("Height", &image_height, 90, 1080);
        changed_local |= ImGui::SliderFloat("FOV", (float*)&camera_vfov, 5.0f, 60.0f);
        if (ImGui::Combo("Kernel", &kernel_choice, "simple\0antialias\0cool\0")) changed_local = true;

        if (changed_local) {
            camera_changed = true;
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
        }

        if (ImGui::Button("Renders")) {
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
        }
        ImGui::SameLine();
        if (ImGui::Button("Save PNG")) {
            stbi_write_png("image.png", image_width, image_height, 3,
                           renderer.get_host_image(), image_width * 3);
        }

        if(ImGui::Button("Reset")) {
            image_width = 1080;
            image_height = 720;
            samples_per_pixel = 1;
            kernel_choice = 2;  
            camera_vfov = 20.0;
            camera_lookat = Point(
                0.5 * (min_pt.m_x + max_pt.m_x),
                0.5 * (min_pt.m_y + max_pt.m_y),
                0.5 * (min_pt.m_z + max_pt.m_z)
            );
            camera_radius = (float)(max_extent * 1.5);
            mouse_x = 0.0f;
            mouse_y = 20.0f;
            
            renderer.resize(image_width, image_height, samples_per_pixel);
            renderer.update_camera(camera_lookfrom, camera_lookat, camera_vfov);
            renderer.render_frame(kernel_choice);
            
        }

        ImGui::End();

        if (renderer.is_ready()) {
            renderer.update_texture();
        }

        // Render to window
        ImGui::Render();
        int dw, dh;
        glfwGetFramebufferSize(window, &dw, &dh);
        glViewport(0, 0, dw, dh);
        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        glMatrixMode(GL_PROJECTION);
        glLoadIdentity();
        glOrtho(0, dw, dh, 0, -1, 1);
        glMatrixMode(GL_MODELVIEW);
        glLoadIdentity();

        glEnable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, renderer.get_texture_id());

        float window_aspect = (float)dw / dh;
        float image_aspect  = (float)image_width / image_height;
        float quad_width, quad_height;
        float x_offset, y_offset;

        if (window_aspect > image_aspect) {
            quad_height = (float)dh;
            quad_width = quad_height * image_aspect;
            x_offset = (dw - quad_width) * 0.5f;
            y_offset = 0;
        } else {
            quad_width = (float)dw;
            quad_height = quad_width / image_aspect;
            x_offset = 0;
            y_offset = (dh - quad_height) * 0.5f;
        }

        glBegin(GL_QUADS);
        glTexCoord2f(0.0f, 1.0f); glVertex2f(x_offset, y_offset + quad_height);
        glTexCoord2f(1.0f, 1.0f); glVertex2f(x_offset + quad_width, y_offset + quad_height);
        glTexCoord2f(1.0f, 0.0f); glVertex2f(x_offset + quad_width, y_offset);
        glTexCoord2f(0.0f, 0.0f); glVertex2f(x_offset, y_offset);
        glEnd();

        glDisable(GL_TEXTURE_2D);
        glBindTexture(GL_TEXTURE_2D, 0);

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        glfwSwapBuffers(window);
    }

    // Cleanup
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}
